#include "hip/hip_runtime.h"
//
// Created by graphics on 12/01/19.
//

#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include "advect_dev.cuh"
#include "sim_pls_cuda_dev.cuh"
#include "cuda_errorcheck.hpp"

__global__ void
advect_particles_3D(int n_particles, scalar_t dt, CUVEC::Vec3d *p, scalar_t *vx, scalar_t *vy, scalar_t *vz, SimParams *C) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n_particles) { // Will cause some thread divergence in the last warp.
        CUVEC::Vec3d vel_start = cu_vel_trilerp(p[i], vx, vy, vz, *C);
        CUVEC::Vec3d pos_mid = p[i] + 0.5 * dt * vel_start;
        CUVEC::Vec3d vel_mid = cu_vel_trilerp(pos_mid, vx, vy, vz, *C);
        p[i] += dt * vel_mid;
//        CUVEC::Vec3d min_coord(0, 0, 0);
//        CUVEC::Vec3d max_coord(C->sim_w, C->sim_h, C->sim_d);
//        p[i] = CUVEC::clamp(p[i], min_coord, max_coord);
    }
}

__host__ void
advect_particles_on_device(int n_blocks, int threads_in_block, int n_particles, scalar_t dt, CUVEC::Vec3d *DEV_p,
                           std::array<scalar_t*, 3> DEV_V, SimParams *C) {
    advect_particles_3D <<< n_blocks, threads_in_block >>> (n_particles, dt, DEV_p, DEV_V[0], DEV_V[1], DEV_V[2], C);
    cuda_check(hipPeekAtLastError());
}

__device__ void get_neighbours_cerped_pos(const CUVEC::Vec3d &pos, scalar_t *Q, scalar_t *sides, SimParams &C) {
    bool clamp = true;
    sides[0] = cu_grid_tricerp({pos[0] + C.dx, pos[1], pos[2]}, Q, clamp, C);
    sides[1] = cu_grid_tricerp({pos[0] - C.dx, pos[1], pos[2]}, Q, clamp, C);
    sides[2] = cu_grid_tricerp({pos[0], pos[1] + C.dx, pos[2]}, Q, clamp, C);
    sides[3] = cu_grid_tricerp({pos[0], pos[1] - C.dx, pos[2]}, Q, clamp, C);
    sides[4] = cu_grid_tricerp({pos[0], pos[1], pos[2] + C.dx}, Q, clamp, C);
    sides[5] = cu_grid_tricerp({pos[0], pos[1], pos[2] - C.dx}, Q, clamp, C);
}

__device__ void get_neighbours_lerped_pos(const CUVEC::Vec3d &pos, scalar_t *Q, scalar_t *sides, SimParams &C) {
    sides[0] = cu_grid_trilerp({pos[0] + C.dx, pos[1], pos[2]}, Q, C, {0, 0, 0});
    sides[1] = cu_grid_trilerp({pos[0] - C.dx, pos[1], pos[2]}, Q, C, {0, 0, 0});
    sides[2] = cu_grid_trilerp({pos[0], pos[1] + C.dx, pos[2]}, Q, C, {0, 0, 0});
    sides[3] = cu_grid_trilerp({pos[0], pos[1] - C.dx, pos[2]}, Q, C, {0, 0, 0});
    sides[4] = cu_grid_trilerp({pos[0], pos[1], pos[2] + C.dx}, Q, C, {0, 0, 0});
    sides[5] = cu_grid_trilerp({pos[0], pos[1], pos[2] - C.dx}, Q, C, {0, 0, 0});
}

__device__ CUVEC::Vec3d
cu_get_grad_lerped_pos(const CUVEC::Vec3d &pos, scalar_t *LS, SimParams &C) {
    CUVEC::Vec3d grad;
    scalar_t sides[6];
    get_neighbours_lerped_pos(pos, LS, sides, C);

    grad[0] = (sides[0] - sides[1]) / (2.0 * C.dx);
    grad[1] = (sides[2] - sides[3]) / (2.0 * C.dx);
    grad[2] = (sides[4] - sides[5]) / (2.0 * C.dx);

    return grad;
}

__device__ CUVEC::Vec3d
cu_get_grad_lerped(const CUVEC::Vec3d &pos, scalar_t *LS_grad_x, scalar_t *LS_grad_y, scalar_t *LS_grad_z,
                   SimParams &C) {
    CUVEC::Vec3d grad;
    grad[0] = cu_grid_trilerp(pos, LS_grad_x, C);
    grad[1] = cu_grid_trilerp(pos, LS_grad_y, C);
    grad[2] = cu_grid_trilerp(pos, LS_grad_z, C);

    return grad;
}

__device__ CUVEC::Vec3d
get_normal(const CUVEC::Vec3d &pos, scalar_t *LS, scalar_t *LS_grad_x, scalar_t *LS_grad_y, scalar_t *LS_grad_z,
           SimParams &C) {
    CUVEC::Vec3d grad;

//    grad = cu_get_grad_lerped(pos, LS_grad_x, LS_grad_y, LS_grad_z, scale_w);
    grad = cu_get_grad_lerped_pos(pos, LS, C);
    scalar_t magnitude = CUVEC::mag(grad);
    if (magnitude > 1e-10) {
        CUVEC::normalize(grad);
        return grad;
    } else {
        return {0, 0, 0};
    }
}

__device__ bool
get_surface_point(const CUVEC::Vec3d &pos, CUVEC::Vec3d &result, scalar_t *LS, scalar_t *LS_grad_x, scalar_t *LS_grad_y,
                  scalar_t *LS_grad_z, SimParams &C) {
    CUVEC::Vec3d search_pt = pos;
    bool clamp = false;
//    scalar_t dist = cu_grid_tricerp(search_pt, LS, clamp, C);
    scalar_t dist = cu_grid_trilerp(search_pt, LS, C);
    scalar_t tol = 1E-12;
    int iters = 0;
    while (fabs(dist) > tol * C.dx && iters < 300) {
        CUVEC::Vec3d normal = get_normal(search_pt, LS, LS_grad_x, LS_grad_y, LS_grad_z, C);
        search_pt -= dist * normal;
        dist = cu_grid_tricerp(search_pt, LS, clamp, C);
//        dist = cu_grid_trilerp(search_pt, LS, C);
        ++iters;
    }
    result = search_pt;
    bool valid = fabs(dist) <= tol * C.dx;
    return valid;
}

__device__ inline unsigned int randhash(unsigned int seed) {
    unsigned int i = (seed ^ 0xA3C59AC3u) * 2654435769u;
    i ^= (i >> 16);
    i *= 2654435769u;
    i ^= (i >> 16);
    i *= 2654435769u;
    return i;
}

__device__ inline float randhashf(unsigned int seed, float a, float b) {
    return ((b - a) * randhash(seed) / (float) UINT_MAX + a);
}

__constant__ scalar_t dir[2] = {-1, 1}; // avoid thread divergence
__global__ void reseed_particles_surface(int n_cells,
                                         CUVEC::Vec3i *coords,
                                         CUVEC::Vec3d *sp,
                                         CUVEC::Vec3d *pp,
                                         CUVEC::Vec3d *np,
                                         scalar_t *LS,
                                         scalar_t *LS_grad_x,
                                         scalar_t *LS_grad_y,
                                         scalar_t *LS_grad_z,
                                         int sp_per_cell,
                                         int sign_per_cell,
                                         scalar_t bandwidth,
                                         SimParams *C
                                        ) {
    (void) sign_per_cell; // Not used because we assign a signed particle with a surface particle each time.
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int cell_id = id / sp_per_cell;

    if (cell_id < n_cells) {
        unsigned int i = coords[cell_id][0];
        unsigned int j = coords[cell_id][1];
        unsigned int k = coords[cell_id][2];

        unsigned int seed = 3 * id;
        CUVEC::Vec3d start(i * C->dx, j * C->dx, k * C->dx);
        scalar_t x_off = C->dx * randhashf(seed++, 0, 1) - 0.5 * C->dx;
        scalar_t y_off = C->dx * randhashf(seed++, 0, 1) - 0.5 * C->dx;
        scalar_t z_off = C->dx * randhashf(seed++, 0, 1) - 0.5 * C->dx;
        CUVEC::Vec3d offset(x_off, y_off, z_off);
        CUVEC::Vec3d newpt = start + offset;
        CUVEC::Vec3d min_coord(0, 0, 0);
        CUVEC::Vec3d max_coord(C->sim_w, C->sim_h, C->sim_d);
        newpt = CUVEC::clamp(newpt, min_coord, max_coord);
//    scalar_t LS_val = cu_grid_trilerp(newpt, LS, *C);
        unsigned int p_id = id;
//    if (fabs(LS_val) < 2.0*bandwidth * C->dx) {
//        if (LS_val > 0) {
//            pp[p_id] = newpt;
//            np[p_id] = CUVEC::Vec3d(-1000, -1000, -1000);
//        } else {
//            np[p_id] = newpt;
//            pp[p_id] = CUVEC::Vec3d(-1000, -1000, -1000);
//        }
        CUVEC::Vec3d surf;
        bool success = get_surface_point(newpt, surf, LS, LS_grad_x, LS_grad_y, LS_grad_z, *C);
//        if (success && (surf[0] < 0 || surf[1] < 0 || surf[2] < 0)){
//            printf("Arrived at a negative value, oops\n");
//        }
        if (success) {
//            surf[0] = fmax(fmin(surf[0], C->sim_w), 0);
//            surf[1] = fmax(fmin(surf[1], C->sim_h), 0);
//            surf[2] = fmax(fmin(surf[2], C->sim_d), 0);
            sp[p_id] = surf;
            // half of signed particles in each direction
            CUVEC::Vec3d p0 =
                    surf + dir[p_id % 2] * 0.25 * C->dx * get_normal(surf, LS, LS_grad_x, LS_grad_y, LS_grad_z, *C);
//            CUVEC::Vec3d p1 = surf - 0.25*C->dx*get_normal(surf, LS, LS_grad_x, LS_grad_y, LS_grad_z, *C);
            np[p_id] = CUVEC::Vec3d(-1000, -1000, -1000);
            pp[p_id] = CUVEC::Vec3d(-1000, -1000, -1000);
            if (cu_grid_trilerp(p0, LS, *C) > 0) {
                pp[p_id] = p0;
            } else {
                np[p_id] = p0;
            }

//            if ( cu_grid_trilerp(p1, LS, *C) > 0 ){
//                pp[p_id] = p1;
//            } else {
//                np[p_id] = p1;
//            }
        } else {
//            printf("Failed to seed a valid start position!\n");
            sp[p_id] = CUVEC::Vec3d(-1000, -1000, -1000);
            np[p_id] = CUVEC::Vec3d(-1000, -1000, -1000);
            pp[p_id] = CUVEC::Vec3d(-1000, -1000, -1000);
        }
//    } else {
//        sp[p_id] = CUVEC::Vec3d(-1000, -1000, -1000);
//        pp[p_id] = CUVEC::Vec3d(-1000, -1000, -1000);
//        np[p_id] = CUVEC::Vec3d(-1000, -1000, -1000);
//    }
    }
}

__host__ void reseed_surface_particles_on_device(int n_cells,
                                                 CUVEC::Vec3i *coords,
                                                 CUVEC::Vec3d *sp,
                                                 CUVEC::Vec3d *pp,
                                                 CUVEC::Vec3d *np,
                                                 scalar_t *LS,
                                                 scalar_t *LS_grad_x,
                                                 scalar_t *LS_grad_y,
                                                 scalar_t *LS_grad_z,
                                                 int sp_per_cell,
                                                 int sign_per_cell,
                                                 scalar_t bandwidth,
                                                 SimParams *C,
                                                 int threads_in_block) {
    int n_blocks = (n_cells*sp_per_cell)/threads_in_block + 1;
    reseed_particles_surface <<< n_blocks, threads_in_block >>>
                                           (n_cells, coords, sp, pp, np, LS, LS_grad_x, LS_grad_y, LS_grad_z,
                                                   sp_per_cell, sign_per_cell, bandwidth, C);
}

__global__ void calculate_particle_cells(int n_p, CUVEC::Vec3d *p, int *cell_ids, SimParams *C) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < n_p) {
        int i = lrint(p[id][0] / C->dx);
        int j = lrint(p[id][1] / C->dx);
        int k = lrint(p[id][2] / C->dx);

//        if (i < 0 || j < 0) {
//            printf("%d, %d, %d, %f, %f, %f\n", i, j, k, p[id][0], p[id][1], p[id][2]);
//        }
        i = max(min(i, C->grid_w-2), 1);
        j = max(min(j, C->grid_h-2), 1);
        k = max(min(k, C->grid_d-2), 1);
        int cell_id = ccti(i, j, k, C->grid_w, C->grid_h);

//        printf("%d \n", cell_id);
        cell_ids[id] = cell_id;
    }
}

__host__ void
calculate_particle_cells_on_device(int n_blocks, int threads_in_block, int n_p, CUVEC::Vec3d *p, int *cell_ids,
                                   SimParams *C) {
    calculate_particle_cells <<< n_blocks, threads_in_block >>> (n_p, p, cell_ids, C);
    cuda_check(hipPeekAtLastError());
}

__host__ void sort_particles_by_key_on_device(CUVEC::Vec3d *p, int *cell_ids, int n_p) {
    // GPU implementation
    thrust::device_ptr<CUVEC::Vec3d> vals(p);
    thrust::device_ptr<int> keys(cell_ids);
    thrust::sort_by_key(keys, keys + n_p, vals);
}

__device__ scalar_t
find_closest_particle(CUVEC::Vec3d *p, int n_p, CUVEC::Vec3d grid_pos, int *cp){
    scalar_t best_dist = 2000;
    for (int i = 0; i<n_p; i++){
        CUVEC::Vec3d sep = p[i] - grid_pos;
        scalar_t dist = CUVEC::mag(sep);
        if (dist<best_dist){
            best_dist = dist;
            *cp = i;
        }
    }

    return best_dist;
}

__device__ void do_grid_redistance_now(scalar_t * LS, CUVEC::Vec3d * p, int * count, CUVEC::Vec3d grid_pos, int p_id, int id, int *cp){
    int n_p = count[id];
    int candidate_cp;
    scalar_t dist = find_closest_particle(&p[p_id], n_p, grid_pos, &candidate_cp);
    if (dist < *LS){
        *LS = dist;
        *cp = candidate_cp + p_id;
    }
}

__device__ int cusgn(scalar_t val) {
    return (val > 0) - (val < 0);
}

// TODO: Improve performance by using the levelset to determine if its worth even looking for a particle nearby.
__global__ void
assign_grid_particle_dist(int n_cells, scalar_t *LS, CUVEC::Vec3d *p, int *index, int *count, int *cp, SimParams *C) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // grid index
    int i = id % C->grid_w;
    int j = (id / C->grid_w) % C->grid_h;
    int k = id / (C->grid_h * C->grid_w);
    CUVEC::Vec3d grid_pos(i * C->dx, j * C->dx, k * C->dx);
//    printf("Processing cell %d, %d, %d\n", i, j, k);
    if (id < n_cells) {
        LS[id] = 1000;
        int p_id = index[id];
        if (p_id == -1) {
            for (int kn = k - 1; kn <= k + 1; kn++) {
                for (int jn = j - 1; jn <= j + 1; jn++) {
                    for (int in = i - 1; in <= i + 1; in++) {
                        if (kn >= 0 && kn < C->grid_d && jn >= 0 && jn < C->grid_h && in >= 0 && in < C->grid_w) {
                            int id_adj = ccti(in, jn, kn, C->grid_w, C->grid_h);
                            p_id = index[id_adj];
                            if (p_id != -1) {
//                                printf("Redistancing with other cell's particles.\n");
                                do_grid_redistance_now(&LS[id], p, count, grid_pos, p_id, id_adj, &cp[id]);
                            }
                        }
                    }
                }
            }
        } else {
//            printf("Redistancing with own cell's particles.\n");
            do_grid_redistance_now(&LS[id], p, count, grid_pos, p_id, id, &cp[id]);
        }
    }
}

__host__ void update_levelset_distances_on_device(int n_cells, int n_blocks, int threads_per_block,
                                                  scalar_t *LS, CUVEC::Vec3d *p, int *index, int *count, int *cp, SimParams* C) {
    assign_grid_particle_dist <<< n_blocks, threads_per_block >>> (n_cells, LS, p, index, count, cp, C);
    cuda_check(hipPeekAtLastError());
}

__global__ void update_LS_signs(int n_cells, scalar_t *LS, scalar_t *LS_p, scalar_t *LS_n) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < n_cells) {
        if (LS_p[id] > LS_n[id]) {
            LS[id] = fabs(LS[id]);
        } else {
            LS[id] = -fabs(LS[id]);
        }
    }
}

//__global__ void generate_curvature(int n_cells, CUVEC::Vec3d *p, int* index, int* count, scalar_t* curv){
//    int id = blockIC.dx.x*blockDim.x + threadIC.dx.x;
//    int i = id % C.grid_w;
//    int j = (id / C.grid_w) % C.grid_h;
//    int k = id / (C.grid_w * C.C.grid_h);
//    CUVEC::Vec3d pos(i*scale_w, j*scale_h, k*scale_d); // cell position
//
//    if (id < n_cells){
//        for(int i = 0; i < count[id]; i++){
//            int p_id = index[id];
//            if (p_id != -1){
//
//            }
//        }
//    }
//}
//
//__host__ void generate_curvature_on_device(int n_cells, int n_blocks, int threads_per_block, CUVEC::Vec3d *p, int* index, int* count, scalar_t* curv){
//
//}